#include <algorithm>
#include <cstdint>
#include <iostream>
#include <filesystem>
#include "utils/parser.hpp"
#include "structures/matrix.hpp"
#include "utils/utils.hpp"
#include "operations/cpu_matrix_vec.hpp"
#include "profiler/profiler.hpp"
#include "operations/gpu_matrix_vec.cuh"
#include "utils/sort_matrix_parallel.cuh"
#include "utils/cuda_utils.cuh"

Mode executionMode = Mode_::GPU;

typedef uint32_t indexType_t;
typedef double dataType_t;

int main(int argc, char **argv) {
  ScopeProfiler prof("main");
  if (argc != 2) {
    std::cerr << "Usage: ./gpu_csr <path_to_mtx_file>" << std::endl;
    exit(1);
  }

  if (!std::filesystem::is_regular_file(argv[1])) {
    std::cerr << argv[1] << " is not a file" << std::endl;
    exit(2);
  }

  std::cout << "GPU-CSR" << std::endl;

  auto retMatrix =
      Utils::parseMatrixMarketFile<indexType_t, dataType_t>(argv[1]);

  if (!retMatrix.has_value()) {
    std::cerr << retMatrix.error() << std::endl;
    exit(3);
  }

  Utils::parallelSort(retMatrix.value());
  retMatrix.value().computeCSR();

  Matrix<indexType_t, dataType_t> matrix = std::move(retMatrix.value());

  std::cout << "matrix csr: " << matrix.csr[matrix.N_ROWS] << std::endl;
  // for (int i = 0; i < matrix.N_ROWS + 1; i++) {
  //   std::cout << matrix.csr[i] << " ";
  // }

  Matrix<indexType_t, dataType_t> vec(MatrixType_::array, matrix.N_ELEM);
  for (int i = 0; i < matrix.N_ELEM; i++) {
    vec.values[i] = 1;
  }

  // std::cout << matrix << std::endl;
  //
  // std::cout << "start vec" << std::endl;
  // std::cout << vec;

  Matrix<indexType_t, dataType_t> resMat(MatrixType_::array, matrix.N_ROWS);
  const indexType_t N_BLOCKS = COMPUTE_N_BLOCKS(indexType_t, matrix.N_ROWS);
  const indexType_t N_THREAD = COMPUTE_N_THREAD(indexType_t, matrix.N_ROWS);

  indexType_t *csr, *columns;
  dataType_t *values, *array, *res1, *res2;
  // GPU allocation
  CUDA_CHECK(hipMalloc(&csr, (matrix.N_ROWS + 1) * sizeof(indexType_t)));
  CUDA_CHECK(hipMalloc(&columns, matrix.N_ELEM * sizeof(indexType_t)));
  CUDA_CHECK(hipMalloc(&values, matrix.N_ELEM * sizeof(dataType_t)));
  CUDA_CHECK(hipMalloc(&array, matrix.N_ELEM * sizeof(dataType_t)));

  res1 = (dataType_t *)calloc(matrix.N_ROWS, sizeof(dataType_t));
  if (!res1) {
    std::cerr << "Calloc error on res1!" << std::endl;
    return EXIT_FAILURE;
  }

  CUDA_CHECK(hipMalloc(&res2, matrix.N_ROWS * sizeof(dataType_t)));

  // GPU copy
  CUDA_CHECK(hipMemcpy(csr, matrix.csr,
                        (matrix.N_ROWS + 1) * sizeof(indexType_t),
                        hipMemcpyHostToDevice));
  CUDA_CHECK(hipMemcpy(columns, matrix.columns,
                        matrix.N_ELEM * sizeof(indexType_t),
                        hipMemcpyHostToDevice));
  CUDA_CHECK(hipMemcpy(values, matrix.values,
                        matrix.N_ELEM * sizeof(dataType_t),
                        hipMemcpyHostToDevice));
  CUDA_CHECK(hipMemcpy(array, vec.values, matrix.N_ELEM * sizeof(dataType_t),
                        hipMemcpyHostToDevice));
  CUDA_CHECK(hipMemcpy(res2, res1, matrix.N_ROWS * sizeof(dataType_t),
                        hipMemcpyHostToDevice));

  std::cout << "Completed all the CUDA malloc and memcpy correctly!"
            << std::endl;
  {
    ScopeProfiler pMult("multiplication");
    // Operations::parallelMultiplicationThreadPerRow<<<N_BLOCKS, N_THREAD>>>(
    //     (indexType_t) matrix.N_ROWS, csr, columns, values, array, res2);
    
    Operations::parallelMultiplicationElementWise<<<N_BLOCKS, N_THREAD>>>(
        (indexType_t) matrix.N_ROWS, csr, columns, values, array, res2);
    hipDeviceSynchronize();
  }

  hipMemcpy(resMat.values, res2, (matrix.N_ROWS) * sizeof(dataType_t),
             hipMemcpyDeviceToHost);

  // std::cout << "res: " << std::endl;
  // for (int i = 0; i < matrix.N_ROWS; i++) {
  //   std::cout << resMat.values[i] << " ";
  // }
  // std::cout << std::endl;

  std::cout << "save: " << std::endl;
  Utils::saveResultsToFile(matrix, vec, resMat);
  CUDA_CHECK(hipFree(csr));
  CUDA_CHECK(hipFree(columns));
  CUDA_CHECK(hipFree(values));
  CUDA_CHECK(hipFree(array));
  CUDA_CHECK(hipFree(res2));
  free(res1);
  return 0;
}
